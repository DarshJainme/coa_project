// speedups_with_cuda.cu
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <omp.h>
#include <vector>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void laplacianKernel(float* input, float* output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        output[i * N + j] = 0.25f * (2.0 * input[(i-1) * N + j] + 2.0 * input[(i+1) * N + j] + 
                                        2.0 * input[i * N + (j-1)] + 2.0 * input[i * N + (j+1)] + 
                                        input[(i-1) * N + (j-1)] + input[(i+1) * N + (j+1)] + 
                                        input[(i-1) * N + (j+1)] + input[(i+1) * N + (j-1)] -
                                        12.0 * input[i * N + j]);
    }
}

int read_time(float& time, string file_name) {
    ifstream infile("seq_execution_time.txt");
    if (!infile.is_open())
    {
        cerr << "Error: Could not open 'execution_time.txt' for reading.\n";
        return 1;
    }

    cout << "Reading execution times from 'execution_time.txt':\n";
    string line;
    while (getline(infile, line))
    {
        time = stof(line);
    }

    infile.close();
    return 0;
}

int main() {
    // Testing CUDA speedup for 10k x 10k matrix
    int N = 10000;
    srand(time(NULL));
    float* input = new float[N * N];
    float* output = new float[N * N];

    // Initialize input with random values
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            input[i*N + j] = rand();
        }
    }

    float *d_matrix, *d_result;
    hipMalloc((void**)&d_matrix, N * N * sizeof(float));
    hipMalloc((void**)&d_result, N * N * sizeof(float));

    hipMemcpy(d_matrix, input, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch kernel
    laplacianKernel<<<numBlocks, threadsPerBlock>>>(d_matrix, d_result, N);

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float cuda_time;
    hipEventElapsedTime(&cuda_time, start, stop);

    float sequential_time;
    read_time(sequential_time, "execution_time.txt");

    // Output the timing output
    cout << "Sequential execution time is: " << sequential_time << " seconds\n\n";
    cout << "Execution time with CUDA: " << cuda_time << " ms" << endl;
    cout << "Speedup with CUDA: " << sequential_time * 1000 / cuda_time << "\n\n";

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(output, d_result, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_matrix);
    hipFree(d_result);

    delete[] input;
    delete[] output;

    return 0;
}
